
#include <hip/hip_runtime.h>

#if GOOGLE_CUDA

#include "ew_op_gpu.h"
//#include <stdio.h>

template <typename T, typename V, uint THREADS, uint WIDTH>
__global__ void __launch_bounds__(THREADS) layer_norm_moments1_CN(
          V*              P1,
          V*              P2,
    const T* __restrict__ X,
    uint K, uint N)
{
    // Stripe the reduction lines with tid and block_n
    uint tid      = threadIdx.x;
    uint block_n  = blockIdx.x;
    uint block_k  = blockIdx.y;

    uint warps = THREADS / 32;
    uint lines = THREADS / WIDTH;
    uint line  = tid     / WIDTH;

    uint n = block_n*WIDTH + (tid % WIDTH);
    uint k = block_k * lines + line;

    uint kn = k*N + n;
    bool bn = n < N;

    uint inc_k  = gridDim.y * lines;
    uint inc_kn = inc_k*N;

    V mean1, mean2;
    ew_zero(mean1);
    ew_zero(mean2);
    #pragma unroll 1
    while (k < K)
    {
        V x = load(add_ptr_u(X, kn), 0, bn);

        mean1 = ew_add(mean1, x);
        mean2 = ew_add(mean2, ew_sqr(x));
        kn += inc_kn;
        k  += inc_k;
    }
    __shared__ V sMean1[THREADS];
    __shared__ V sMean2[THREADS];

    sMean1[tid] = mean1;
    sMean2[tid] = mean2;

    __syncthreads();

    if (tid < 32)
    {
        for (uint i = 1; i < warps; i++)
            mean1 = ew_add(mean1, sMean1[tid + i*32]);

        // if the line width is less than a warp, reduce the lines within a warp
        for (int i = 16; i >= WIDTH; i >>= 1)
            mean1 = ew_warp_sum(mean1, i);

        // output a partial sums
        if (tid < WIDTH && bn)
            store(add_ptr_u(P1, block_k*N + n), mean1);
    }
    else if (tid < 64)
    {
        tid -= 32;
        mean2 = ew_add(mean2, sMean2[tid + 0*32]);
        for (uint i = 2; i < warps; i++)
            mean2 = ew_add(mean2, sMean2[tid + i*32]);

        // if the line width is less than a warp, reduce the lines within a warp
        for (int i = 16; i >= WIDTH; i >>= 1)
            mean2 = ew_warp_sum(mean2, i);

        // output a partial sums
        if (tid < WIDTH && bn)
            store(add_ptr_u(P2, block_k*N + n), mean2);
    }
}

// Reduce partial sums
__global__ void __launch_bounds__(256) layer_norm_moments2_CN(
          float*              Mean,
          float*              Rstd,
    const float* __restrict__ P1,
    const float* __restrict__ P2,
    uint nPartials, uint N, float rcpK, float epsilon)
{
    uint tid = threadIdx.x;
    uint bid = blockIdx.x;

    // load in 8 units of n wide to allow efficient transpose in L1 cache
    uint n = bid*8 + tid/32;
    uint k = tid & 31;

    uint kn = k*N + n;
    bool bn = n < N;

    // force compute outside of loop
    asm("mov.b32 %0, %0;" : "+r"(kn) : );

    float mean1 = 0.0f, mean2 = 0.0f;
    // We should generally have #SMs * 2 partials.
    #pragma unroll 1
    while (k < nPartials)
    {
#if __CUDA_ARCH__ >= 700
        const int UNROLL = 5; // 2*80 partials
#else
        const int UNROLL = 4; // 2*56 partials
#endif

        bool bnk[UNROLL];
        bnk[0] = bn;
        for (int i = 1; i < UNROLL; i++)
            bnk[i] = bn && (k+32*i < nPartials);

        for (int i = 0; i < UNROLL; i++)
        {
            mean1 += load(add_ptr_u(P1, kn + N*32*i), 0, bnk[i]);
            mean2 += load(add_ptr_u(P2, kn + N*32*i), 0, bnk[i]);
        }
        kn += 32*UNROLL*N;
        k  += 32*UNROLL;
    }
    for (uint i = 16; i > 0; i >>= 1)
    {
        mean1 += shfl_xor(mean1, i);
        mean2 += shfl_xor(mean2, i);
    }
    if (bn & (tid & 31) == 0)
    {
        // var  = mean(x**2) - mean(x)**2
        // rstd = 1/sqrt(var)
        mean1 *= rcpK;
        mean2 *= rcpK;
        float rstd = rsqrtf(precise_sub(mean2, ew_sqr(mean1)) + epsilon);
        store(add_ptr_u(Mean, n), mean1);
        store(add_ptr_u(Rstd, n), rstd);
    }
}

// xstdr = rcp(sqrt(xvar + epsilon))
// xhat  = xmean * xstdr
// y     = xhat*g + b
template <typename T, int UNROLL>
__global__ void __launch_bounds__(32) layer_norm_CN(
               T*              Y,
    const      T* __restrict__ X,
    const float4* __restrict__ Mean,
    const float4* __restrict__ Rstd,
    const  float* __restrict__ G,
    const  float* __restrict__ B,
    int K, int N, int relu)
{
    __shared__ float Gain[UNROLL*2];
    __shared__ float Bias[UNROLL*2];

    int tid   = threadIdx.x;
    int idx_K = blockIdx.x * UNROLL*2;
    int idx_N = blockIdx.y * 16;

    // load gain/bias for this K-block
    int ki = idx_K + tid;
    if (tid < UNROLL*2 && ki < K)
    {
        Gain[tid] = G[ki];
        Bias[tid] = B[ki];
    }

    int tid16 = tid >> 4;
    int tid15 = tid & 15;

    int k = idx_K + tid16;
    int n = idx_N + tid15;

    bool bn = n < N;

    int xi  = k*N + n;
    int inc = N * 2;

    float4 rstd = load(Rstd, n, bn);
    float4 mean = load(Mean, n, bn);

    #pragma unroll
    for (int j = 0; j < UNROLL; j++)
    {
        bool bnk = bn && k < K;
        float4 x = load(X, xi, bnk);

        float g = Gain[tid16];
        float b = Bias[tid16];

        // xhat = (x - mean) / sqrt(var + epsilon)
        //   y  = g * xhat + b
        float4 xhat = ew_mul(ew_sub(x, mean), rstd);
        float4    y = ew_add(ew_mul(xhat, g), b);

        if (relu)
            y = ew_relu(y);

        store(Y, y, xi, bnk);

        k     += 2;
        tid16 += 2;
        xi    += inc;
    }
}
template <typename T, typename V>
bool LayerNormForward_CN(CUstream stream, int SMs,
              T* y,
          float* mean,
          float* rstd,
          float* p1,
          float* p2,
    const     T* x,
    const float* g,
    const float* b,
    float epsilon, int K, int N, float rcpK, int relu)
{
    const      V*    X = (const V*)x;
    const float4* Mean = (const float4*)mean;
    const float4* Rstd = (const float4*)rstd;
          float4*   P1 = (      float4*)p1;
          float4*   P2 = (      float4*)p2;

    uint gridN64 = (N >> 6) + ((N &  63) != 0);
    uint gridN8  = (N >> 3) + ((N &   7) != 0);
    uint gridK8  = (K >> 3) + ((K &   7) != 0);

    uint nPartials = gridN64 > 1 ? SMs : SMs*2;
    if (K <= 8*nPartials)
        layer_norm_moments1_CN<V,float4,128,16><<<dim3(gridN64, nPartials),128,0,stream>>>(P1, P2, X, K, N>>2);
    else
        layer_norm_moments1_CN<V,float4,256,16><<<dim3(gridN64, nPartials),256,0,stream>>>(P1, P2, X, K, N>>2);

    layer_norm_moments2_CN<<<gridN8,256,0,stream>>>(mean, rstd, p1, p2, nPartials, N, rcpK, epsilon);

    layer_norm_CN<V,4><<<dim3(gridK8, gridN64),32, 0,stream>>>((V*)y, X, Mean, Rstd, g, b, K, N>>2, relu);
    return true; // TODO
}
template bool LayerNormForward_CN<float,float4>(CUstream stream, int SMs, float* y, float* mean, float* rstd, float* p1, float* p2, const float* x, const float* g, const float* b, float epsilon, int K, int N, float rcpK, int relu);
template bool LayerNormForward_CN<ehalf,ehalf4>(CUstream stream, int SMs, ehalf* y, float* mean, float* rstd, float* p1, float* p2, const ehalf* x, const float* g, const float* b, float epsilon, int K, int N, float rcpK, int relu);
template bool LayerNormForward_CN<bhalf,bhalf4>(CUstream stream, int SMs, bhalf* y, float* mean, float* rstd, float* p1, float* p2, const bhalf* x, const float* g, const float* b, float epsilon, int K, int N, float rcpK, int relu);


// dg = sum(dy * xhat(x), axis=1)
// db = sum(dy, axis=1)
template <typename T>
__global__ void __launch_bounds__(128) layer_norm_dg_db_CN(
           float*              DG,
           float*              DB,
    const      T* __restrict__ DY,
    const      T* __restrict__ X,
    const  float* __restrict__ Gain,
    const  float* __restrict__ Bias,
    const float4* __restrict__ Mean,
    const float4* __restrict__ Rstd,
    int K, int N, int relu)
{
    __shared__ float gain[8];
    __shared__ float bias[8];

    int tid   = threadIdx.x;
    int idx_K = blockIdx.x * 8;

    // load gain/bias for this K-block
    int ki = idx_K + tid;
    if (relu && tid < 8 && ki < K)
    {
        gain[tid] = Gain[ki];
        bias[tid] = Bias[ki];
    }
    int tid16 = tid >> 4;
    int tid15 = tid & 15;
    int k     = idx_K + tid16;

    __syncthreads();
    float dg = 0.0f, db = 0.0f;
    if (k < K)
    {
        int N4 = N >> 2;
        int xi = k*N4;
        X  += xi;
        DY += xi;

        float4 dg4, db4;
        ew_zero(dg4);
        ew_zero(db4);
        for (int n = tid15; n < N4; n += 16)
        {
            float4 x    = load(X,    n);
            float4 dy   = load(DY,   n);
            float4 rstd = load(Rstd, n);
            float4 mean = load(Mean, n);

            // xhat = (x - mean) * rstd
            float4 xhat = ew_mul(ew_sub(x, mean), rstd);

            if (relu)
            {
                float g = gain[tid16];
                float b = bias[tid16];
                dy = ew_relu_grad(dy, ew_add(ew_mul(xhat, g), b));
            }

            dg4 = ew_add(ew_mul(dy, xhat), dg4);
            db4 = ew_add(dy, db4);
        }
        dg = ew_sum(dg4);
        db = ew_sum(db4);
    }
    // reduce each half warp
    for (int i = 8; i > 0; i >>= 1)
    {
        dg += shfl_xor(dg, i);
        db += shfl_xor(db, i);
    }
    if (k < K && tid15 == 0)
    {
        DG[k] = dg;
        DB[k] = db;
    }
}

// dy    = dy * g
// sum1  = sum(xhat * dy, axis=0)
// sum2  = sum(dy, axis=0)
template <typename T, uint THREADS, uint WIDTH>
__global__ void __launch_bounds__(THREADS) layer_norm_dx_sum1_CN(
          float4*              P1,
          float4*              P2,
    const      T* __restrict__ DY,
    const      T* __restrict__ X,
    const  float* __restrict__ Gain,
    const  float* __restrict__ Bias,
    const float4* __restrict__ Mean,
    const float4* __restrict__ Rstd,
    int K, int N, int relu)
{
    // Stripe the reduction lines with tid and block_n
    uint tid      = threadIdx.x;
    uint block_n  = blockIdx.x;
    uint block_k  = blockIdx.y;

    uint warps = THREADS / 32;
    uint lines = THREADS / WIDTH;
    uint line  = tid     / WIDTH;

    uint n = block_n*WIDTH + (tid % WIDTH);
    uint k = block_k * lines + line;

    uint kn = k*N + n;
    bool bn = n < N;

    uint inc_k  = gridDim.y * lines;
    uint inc_kn = inc_k*N;

    float4 rstd = load(Rstd, n, bn);
    float4 mean = load(Mean, n, bn);

    float4 sum1, sum2;
    ew_zero(sum1);
    ew_zero(sum2);
    #pragma unroll 1
    while (k < K)
    {
        float4 dy = load(add_ptr_u(DY, kn), 0, bn);
        float4 x  = load(add_ptr_u(X,  kn), 0, bn);

        float gain = load(add_ptr_u(Gain, k), 0, bn);
        float bias = load(add_ptr_u(Bias, k), 0, bn && relu != 0);

        float4 xhat = ew_mul(ew_sub(x, mean), rstd);
        if (relu != 0)
            dy = ew_relu_grad(dy, ew_add(ew_mul(xhat, gain), bias));
        dy = ew_mul(dy, gain);

        sum1 = ew_add(sum1, ew_mul(dy, xhat));
        sum2 = ew_add(sum2, dy);

        kn += inc_kn;
        k  += inc_k;
    }
    __shared__ float4 sSum1[THREADS];
    __shared__ float4 sSum2[THREADS];

    sSum1[tid] = sum1;
    sSum2[tid] = sum2;

    __syncthreads();

    if (tid < 32)
    {
        for (uint i = 1; i < warps; i++)
            sum1 = ew_add(sum1, sSum1[tid + i*32]);

        // if the line width is less than a warp, reduce the lines within a warp
        for (int i = 16; i >= WIDTH; i >>= 1)
            sum1 = ew_warp_sum(sum1, i);

        // output a partial sums
        if (tid < WIDTH && bn)
            store(add_ptr_u(P1, block_k*N + n), sum1);
    }
    else if (tid < 64)
    {
        tid -= 32;
        sum2 = ew_add(sum2, sSum2[tid + 0*32]);
        for (uint i = 2; i < warps; i++)
            sum2 = ew_add(sum2, sSum2[tid + i*32]);

        // if the line width is less than a warp, reduce the lines within a warp
        for (int i = 16; i >= WIDTH; i >>= 1)
            sum2 = ew_warp_sum(sum2, i);

        // output a partial sums
        if (tid < WIDTH && bn)
            store(add_ptr_u(P2, block_k*N + n), sum2);
    }
}
// Reduce partial sums
__global__ void __launch_bounds__(256) layer_norm_dx_sum2_CN(float* Sum1, float* Sum2, uint nPartials, uint N)
{
    uint tid = threadIdx.x;
    uint bid = blockIdx.x;

    // load in 8 units of n wide to allow efficient transpose in L1 cache
    uint n = bid*8 + tid/32;
    uint k = tid & 31;

    float* Sum = Sum1;
    if (n >= N)
    {
        n  -= N;
        Sum = Sum2;
    }
    uint kn = k*N + n;
    bool bn = n < N;

    // force compute outside of loop
    asm("mov.b32 %0, %0;" : "+r"(kn) : );

    float sum = 0.0f;
    // We should generally have #SMs * 2 partials.
    #pragma unroll 1
    while (k < nPartials)
    {
#if __CUDA_ARCH__ >= 700
        const int UNROLL = 5; // 2*80 partials
#else
        const int UNROLL = 4; // 2*56 partials
#endif

        bool bnk[UNROLL];
        bnk[0] = bn;
        for (int i = 1; i < UNROLL; i++)
            bnk[i] = bn && (k+32*i < nPartials);

        for (int i = 0; i < UNROLL; i++)
            sum += load(add_ptr_u((const float*)Sum, kn + N*32*i), 0, bnk[i]);

        kn += 32*UNROLL*N;
        k  += 32*UNROLL;
    }
    for (uint i = 16; i > 0; i >>= 1)
    {
        sum += shfl_xor(sum, i);
    }
    if (bn & (tid & 31) == 0)
        store(add_ptr_u(Sum, n), sum);

}

// dy = dy * g
// dx = (dy - ((xhat * sum1 + sum2) * rcpK)) * xstdr
template <typename T, int UNROLL>
__global__ void __launch_bounds__(32) layer_norm_dx_CN(
               T*              DX,
    const      T* __restrict__ DY,
    const      T* __restrict__ X,
    const  float* __restrict__ Gain,
    const  float* __restrict__ Bias,
    const float4* __restrict__ Mean,
    const float4* __restrict__ Rstd,
    const float4* __restrict__ Sum1,
    const float4* __restrict__ Sum2,
    int K, int N, float rcpK, int relu)
{
    __shared__ float gain[UNROLL*2];
    __shared__ float bias[UNROLL*2];

    int tid   = threadIdx.x;
    int idx_K = blockIdx.x * UNROLL*2;
    int idx_N = blockIdx.y * 16;

    // load gain/bias for this K-block
    int ki = idx_K + tid;
    if (tid < UNROLL*2 && ki < K)
    {
        gain[tid] = Gain[ki];
        bias[tid] = Bias[ki];
    }

    int tid16 = tid >> 4;
    int tid15 = tid & 15;

    int k = idx_K + tid16;
    int n = idx_N + tid15;

    int  N4 = N  >> 2;
    bool bn = n < N4;

    int xi  = k*N4 + n;
    int inc = N4 * 2;

    float4 rstd = load(Rstd, n, bn);
    float4 mean = load(Mean, n, bn);
    float4 sum1 = load(Sum1, n, bn);
    float4 sum2 = load(Sum2, n, bn);

    #pragma unroll 4
    for (int j = 0; j < UNROLL; j++)
    {
        bool bnk = bn && k < K;
        float4  x = load( X, xi, bnk);
        float4 dy = load(DY, xi, bnk);
        float   g = gain[tid16];
        float   b = bias[tid16];

        float4 xhat = ew_mul(ew_sub(x, mean), rstd);
        if (relu)
            dy = ew_relu_grad(dy, ew_add(ew_mul(xhat, g), b));
        dy = ew_mul(dy, g);

        // dx = (dy - ((xhat * sum1 + sum2) * rcpK)) * rstd;
        float4 dx = ew_mul(ew_sub(dy, ew_mul(ew_add(ew_mul(xhat, sum1), sum2), rcpK)), rstd);

        store(DX, dx, xi, bnk);
        k     += 2;
        tid16 += 2;
        xi    += inc;
    }
}

template <typename T, typename V>
bool LayerNormBackward_CN(CUstream stream, int SMs,
              T* dx,
          float* dg,
          float* db,
          float* sum1,
          float* sum2,
    const     T* dy,
    const     T* x,
    const float* g,
    const float* b,
    const float* mean,
    const float* rstd,
    float epsilon, int K, int N, float rcpK, int relu)
{
    uint gridN64 = (N >> 6) + ((N &  63) != 0);
    uint gridN8  = (N >> 3) + ((N &   7) != 0);
    uint gridK8  = (K >> 3) + ((K &   7) != 0);
    uint nPartials = gridN64 > 1 ? SMs : SMs*2;

          V* DX = (      V*)dx;
    const V* DY = (const V*)dy;
    const V*  X = (const V*)x;

    const float4* Mean = (const float4*)mean;
    const float4* Rstd = (const float4*)rstd;
    const float4* Sum1 = (const float4*)sum1;
    const float4* Sum2 = (const float4*)sum2;

    layer_norm_dg_db_CN<V><<<gridK8,128,0,stream>>>(dg, db, DY, X, g, b, Mean, Rstd, K, N, relu);

    if (K <= 8*nPartials)
        layer_norm_dx_sum1_CN<V,128,16><<<dim3(gridN64, nPartials),128,0,stream>>>((float4*)sum1, (float4*)sum2, DY, X, g, b, Mean, Rstd, K, N>>2, relu);
    else
        layer_norm_dx_sum1_CN<V,256,16><<<dim3(gridN64, nPartials),256,0,stream>>>((float4*)sum1, (float4*)sum2, DY, X, g, b, Mean, Rstd, K, N>>2, relu);

    layer_norm_dx_sum2_CN<<<gridN8*2,256,0,stream>>>(sum1, sum2, nPartials, N);

    layer_norm_dx_CN<V,4><<<dim3(gridK8, gridN64),32,0,stream>>>(DX, DY, X, g, b, Mean, Rstd, Sum1, Sum2, K, N, rcpK, relu);

    return true; // TODO
}

template bool LayerNormBackward_CN<float,float4>(CUstream stream, int SMs, float* dx, float* dg, float* db, float* sum1, float* sum2, const float* dy, const float* x, const float* g, const float* b, const float* mean, const float* rstd, float epsilon, int K, int N, float rcpK, int relu);
template bool LayerNormBackward_CN<ehalf,ehalf4>(CUstream stream, int SMs, ehalf* dx, float* dg, float* db, float* sum1, float* sum2, const ehalf* dy, const ehalf* x, const float* g, const float* b, const float* mean, const float* rstd, float epsilon, int K, int N, float rcpK, int relu);
template bool LayerNormBackward_CN<bhalf,bhalf4>(CUstream stream, int SMs, bhalf* dx, float* dg, float* db, float* sum1, float* sum2, const bhalf* dy, const bhalf* x, const float* g, const float* b, const float* mean, const float* rstd, float epsilon, int K, int N, float rcpK, int relu);


// Sparse Projection Code

template <typename T, typename V, int SHFT>
__global__ void __launch_bounds__(128) gather_scatter(
            T*              Z,
    const   T* __restrict__ X,
    const int* __restrict__ Lut,
    int K, int N)
{
    int tid   = threadIdx.x;
    int idx_K = blockIdx.x;
    int idx_N = blockIdx.y;

    int tidK = tid >> SHFT;
    int tidN = tid & ((1<<SHFT)-1);

    int zk = (idx_K << (7-SHFT)) + tidK;
    int  n = (idx_N <<    SHFT)  + tidN;

    if (zk < K && n < N)
    {
        int xk = load(Lut, zk);

        int zi = zk*N + n;
        int xi = xk*N + n;

        V x = load(X, xi, xk >= 0);

        store(Z, x, zi);
    }
}
template <typename T, typename V, int SHFT>
__global__ void __launch_bounds__(128) scatter_add(
            T*              Z, // large tensor
    const   T* __restrict__ X, // large tensor
    const   T* __restrict__ Y, // small tensor
    const int* __restrict__ Lut,
    int K, int N)
{
    int tid   = threadIdx.x;
    int idx_K = blockIdx.x;
    int idx_N = blockIdx.y;

    int tidK = tid >> SHFT;
    int tidN = tid & ((1<<SHFT)-1);

    int yk = (idx_K << (7-SHFT)) + tidK;
    int  n = (idx_N <<    SHFT) +  tidN;

    if (yk < K && n < N)
    {
        int xk = load(Lut, yk);

        int yi = yk*N + n;
        int xi = xk*N + n;

        V y = load(Y, yi);
        V x = load(X, xi);

        store(Z, ew_add(x, y), xi);
    }
}
template <typename T, typename V, int SHFT>
__global__ void __launch_bounds__(128) scatter_mul(
            T*              Z, // large tensor
    const   T* __restrict__ X, // large tensor
    const   T* __restrict__ Y, // small tensor
    const int* __restrict__ Lut,
    int K, int N)
{
    int tid   = threadIdx.x;
    int idx_K = blockIdx.x;
    int idx_N = blockIdx.y;

    int tidK = tid >> SHFT;
    int tidN = tid & ((1<<SHFT)-1);

    int xk = (idx_K << (7-SHFT)) + tidK;
    int  n = (idx_N <<    SHFT) +  tidN;

    if (xk < K && n < N)
    {
        int yk = load(Lut, xk);

        int xi = xk*N + n;
        int yi = yk*N + n;

        V x = load(X, xi);
        V y = load(Y, yi, yk >= 0);
        V z = yk >= 0 ? ew_mul(x, y) : x; // pass through if unmapped

        store(Z, z, xi);
    }
}
template <typename T, typename V, int SHFT>
__global__ void __launch_bounds__(128) sparse_mul_grad(
            T*              DX, // large tensor
            T*              DY, // small tensor
    const   T* __restrict__ DZ, // large tensor (same pointer as DX)
    const   T* __restrict__ X,  // large tensor
    const   T* __restrict__ Y,  // small tensor
    const int* __restrict__ Lut,
    int K, int N)
{
    int tid   = threadIdx.x;
    int idx_K = blockIdx.x;
    int idx_N = blockIdx.y;

    int tidK = tid >> SHFT;
    int tidN = tid & ((1<<SHFT)-1);

    int yk = (idx_K << (7-SHFT)) + tidK;
    int  n = (idx_N <<    SHFT) +  tidN;

    if (yk < K && n < N)
    {
        int xk = load(Lut, yk);

        int yi = yk*N + n;
        int xi = xk*N + n;

        V y  = load(Y,  yi);
        V x  = load(X,  xi);
        V dz = load(DZ, xi);

        store(DX, ew_mul(dz, y), xi);
        store(DY, ew_mul(dz, x), yi);
    }
}

#define OP_GAT 0
#define OP_SCT 1
#define OP_ADD 2
#define OP_MUL 3

template <typename T, typename V4, typename V8>
bool SparseOp(CUstream stream,
            T* z,
    const   T* x,
    const   T* y,
    const int* lut,
    int op, int K, int N)
{
    int gridN = (N >> 6) + ((N & 63) != 0);

    if (sizeof(T) == 2 && (N & 7) == 0)
    {
              V8* Z = (      V8*)z;
        const V8* X = (const V8*)x;
        const V8* Y = (const V8*)y;

        // blockK = 128 / 8 = 16
        int gridK = (K >> 4) + ((K & 15) != 0);
        dim3 grid(gridK, gridN, 1);
        switch(op)
        {
            case OP_GAT: gather_scatter<V8,float8,3><<<grid,128,0,stream>>>(Z, X,    lut, K, N>>3); break;
            case OP_SCT: gather_scatter<V8,float8,3><<<grid,128,0,stream>>>(Z, X,    lut, K, N>>3); break;
            case OP_ADD:    scatter_add<V8,float8,3><<<grid,128,0,stream>>>(Z, X, Y, lut, K, N>>3); break;
            case OP_MUL:    scatter_mul<V8,float8,3><<<grid,128,0,stream>>>(Z, X, Y, lut, K, N>>3); break;
        }
    }
    else if ((N & 3) == 0)
    {
              V4* Z = (      V4*)z;
        const V4* X = (const V4*)x;
        const V4* Y = (const V4*)y;

        // blockK = 128 / 16 = 8
        int gridK = (K >> 3) + ((K & 7) != 0);
        dim3 grid(gridK, gridN, 1);
        switch(op)
        {
            case OP_GAT: gather_scatter<V4,float4,4><<<grid,128,0,stream>>>(Z, X,    lut, K, N>>2); break;
            case OP_SCT: gather_scatter<V4,float4,4><<<grid,128,0,stream>>>(Z, X,    lut, K, N>>2); break;
            case OP_ADD:    scatter_add<V4,float4,4><<<grid,128,0,stream>>>(Z, X, Y, lut, K, N>>2); break;
            case OP_MUL:    scatter_mul<V4,float4,4><<<grid,128,0,stream>>>(Z, X, Y, lut, K, N>>2); break;
        }
    }
    return true; // TODO
}

template <typename T, typename V4, typename V8>
bool SparseMulGrad(CUstream stream,
            T* dx,
            T* dy,
    const   T* dz,
    const   T* x,
    const   T* y,
    const int* lut,
    int K, int N)
{
    int gridN = (N >> 6) + ((N & 63) != 0);

    if (sizeof(T) == 2 && (N & 7) == 0)
    {
              V8* DX = (      V8*)dx;
              V8* DY = (      V8*)dy;
        const V8* DZ = (const V8*)dz;
        const V8*  X = (const V8*)x;
        const V8*  Y = (const V8*)y;

        // blockK = 128 / 8 = 16
        int gridK = (K >> 4) + ((K & 15) != 0);
        dim3 grid(gridK, gridN, 1);

        sparse_mul_grad<V8,float8,3><<<grid,128,0,stream>>>(DX, DY, DZ, X, Y, lut, K, N>>3);
    }
    else if ((N & 3) == 0)
    {
              V4* DX = (      V4*)dx;
              V4* DY = (      V4*)dy;
        const V4* DZ = (const V4*)dz;
        const V4*  X = (const V4*)x;
        const V4*  Y = (const V4*)y;

        // blockK = 128 / 16 = 8
        int gridK = (K >> 3) + ((K & 7) != 0);
        dim3 grid(gridK, gridN, 1);
        sparse_mul_grad<V4,float4,4><<<grid,128,0,stream>>>(DX, DY, DZ, X, Y, lut, K, N>>2);
    }
    return true; // TODO
}

template bool SparseOp<float,float4,float8>(CUstream stream, float* z, const float* x, const float* y, const int* lut, int op, int K, int N);
template bool SparseOp<ehalf,ehalf4,ehalf8>(CUstream stream, ehalf* z, const ehalf* x, const ehalf* y, const int* lut, int op, int K, int N);
template bool SparseOp<bhalf,bhalf4,bhalf8>(CUstream stream, bhalf* z, const bhalf* x, const bhalf* y, const int* lut, int op, int K, int N);

template bool SparseMulGrad<float,float4,float8>(CUstream stream, float* dx, float* dy, const float* dz, const float* x, const float* y, const int* lut, int K, int N);
template bool SparseMulGrad<ehalf,ehalf4,ehalf8>(CUstream stream, ehalf* dx, ehalf* dy, const ehalf* dz, const ehalf* x, const ehalf* y, const int* lut, int K, int N);
template bool SparseMulGrad<bhalf,bhalf4,bhalf8>(CUstream stream, bhalf* dx, bhalf* dy, const bhalf* dz, const bhalf* x, const bhalf* y, const int* lut, int K, int N);

#endif // GOOGLE_CUDA

// cuobjdump -xelf blocksparse_ops.5.sm_60.cubin blocksparse_ops.so
// cuobjdump -xelf blocksparse_ops.6.sm_61.cubin blocksparse_ops.so

// nvdisasm -c -raw blocksparse_ops.5.sm_60.cubin  > blocksparse_ops.5.sm_60.sass
// nvdisasm -c -raw blocksparse_ops.6.sm_61.cubin  > blocksparse_ops.6.sm_61.sass


